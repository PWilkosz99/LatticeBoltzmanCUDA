
#include <hip/hip_runtime.h>
__constant__ double LATTICE_SPEED = 0.1;
__constant__ double TAU = 0.9;
__constant__ int DIRECTIONS = 9;
__constant__ int DIMENSIONS = 2;

#define KERNEL_HEADER(xvar, yvar, wvar, hvar) \
  const int x = blockIdx.x;\
  const int y = blockIdx.y;\
  const int width = gridDim.x;\
  const int height = gridDim.y
    

__global__ void streaming(double *out, double *in) {
  KERNEL_HEADER(x, y, width, height);

  int dir = 0;
  for(int i = -1; i <= 1; i++) {
    for(int j = -1; j <= 1; j++) {
        int target = y * DIRECTIONS + x * DIRECTIONS * height + dir;
        int xs = x + i;
        int ys = y + j;

        int source = ys * DIRECTIONS + xs * DIRECTIONS * height + dir;
        if(xs >= 0 && xs < width && ys >= 0 && ys < height) 
        {
            out[target] = in[source];
        } 
        else 
        {
            int bounce_dir = (1 - j) + (1 - i) * 3 ;
            int bounce_src = bounce_dir + y * DIRECTIONS + x * DIRECTIONS * height;
            out[target] = in[bounce_src];
        }
        dir++; 
    }
  }
}

__global__ void density(double *out, double* in) {
  KERNEL_HEADER(x, y, width, height);

  int target = y + x * height;
  out[target] = 0;

  int z = 0;
  for(int i = -1; i <= 1; i++) {
    for(int j = -1; j <= 1; j++) {
      int source = z + y * DIRECTIONS + x * DIRECTIONS * height;
      out[target] += in[source];
      z++;
    }
  }
}

__global__ void velocity(double *out, double* density, double* directions) {
  KERNEL_HEADER(x, y, width, height);

  // Compute target indices.
  const int target_x = 0 + y * DIMENSIONS + x * height * DIMENSIONS;
  const int target_y = 1 + y * DIMENSIONS + x * height * DIMENSIONS;
  out[target_x] = 0;
  out[target_y] = 0;

  int z = 0;
  for(int i = -1; i <= 1; i++) {
    for(int j = -1; j <= 1; j++) {
      const int source = z + y * DIRECTIONS + x * DIRECTIONS * height;
      out[target_x] += directions[source] * i;
      out[target_y] += directions[source] * j;
      z++;
    }
  }

  const int target = y + x * height;
  out[target_x] *= LATTICE_SPEED / density[target];
  out[target_y] *= LATTICE_SPEED / density[target];
}

__global__ void equilibrium(double *eq, double* density, double* velocity) {
  KERNEL_HEADER(x, y, width, height);

  const int density_src = y + x * height;
  const int xvel_src = 0 + y * DIMENSIONS + x * height * DIMENSIONS;
  const int yvel_src = 1 + y * DIMENSIONS + x * height * DIMENSIONS;

  // u . u
  double velmag = velocity[xvel_src] * velocity[xvel_src] + velocity[yvel_src] * velocity[yvel_src];

  int z = 0;
  for(int i = -1; i <= 1; i++) {
    for(int j = -1; j <= 1; j++) {
        // Compute the weight.
        double weight;
        if(i == 0 && j == 0) {
            weight = 4.0 / 9.0;
        } else if(i == 0 || j == 0) {
            weight = 1.0 / 9.0;
        } else {
            weight = 1.0 / 36.0;
        }

        // e_i . u
        double dotprod = i * velocity[xvel_src] + j * velocity[yvel_src];

        double sum = 1.0;
        sum += 3 / LATTICE_SPEED * dotprod;
        sum += 4.5 / (LATTICE_SPEED * LATTICE_SPEED) * dotprod * dotprod;
        sum -= 1.5 / (LATTICE_SPEED * LATTICE_SPEED) * velmag;

        const int target = z + y * DIRECTIONS + x * DIRECTIONS * height;
        eq[target] = weight * density[density_src] * sum;

        z++;
    }
  }
}

__global__ void update(double *out, double* equilibrium, double* directions) {
  KERNEL_HEADER(x, y, width, height);

  int z = 0;
  for(int i = -1; i <= 1; i++) {
    for(int j = -1; j <= 1; j++) {
        const int target = z + y * DIRECTIONS + x * DIRECTIONS * height;
        out[target] = directions[target] - (directions[target] - equilibrium[target]) / TAU;
        z++;
    }
  }
}
